/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/

#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

static inline void check(hipError_t err, const char *context)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << context << ": "
                  << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b)
{
    return (a + b - 1) / b;
}

// static inline int roundup(int a, int b)
// {
//     return divup(a, b) * b;
// }

__device__ void foo(int i, int j) {}

__global__ void mykernel(float *result, const float *data, int nx, int ny)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= ny || j >= ny)
    {
        return;
    }
    float dot_product = 0;
    for (int k = 0; k < nx; ++k)
    {
        dot_product += data[k + i * nx] * data[k + j * nx];
    }
    result[i + j * ny] = dot_product;
}

void correlate(int ny, int nx, const float *data, float *result)
{
    float *normalized = new float[ny * nx];

    // Normalize rows to have mean 0
    for (int i = 0; i < ny; ++i)
    {
        float mean = 0;
        for (int j = 0; j < nx; ++j)
        {
            mean += data[j + i * nx];
        }
        mean /= nx;
        for (int j = 0; j < nx; ++j)
        {
            normalized[j + i * nx] = data[j + i * nx] - mean;
        }
    }

    // Normalize rows to have norm of 0
    for (int i = 0; i < ny; ++i)
    {
        float norm = 0;
        for (int j = 0; j < nx; ++j)
        {
            double value = normalized[j + i * nx];
            norm += value * value;
        }
        norm = sqrt(norm);
        for (int j = 0; j < nx; ++j)
        {
            normalized[j + i * nx] /= norm;
        }
    }

    float *dGPU = NULL;
    CHECK(hipMalloc((void **)&dGPU, nx * ny * sizeof(float)));
    float *rGPU = NULL;
    CHECK(hipMalloc((void **)&rGPU, ny * ny * sizeof(float)));
    CHECK(hipMemcpy(dGPU, normalized, nx * ny * sizeof(float), hipMemcpyHostToDevice));
    // Run kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(ny, dimBlock.x), divup(ny, dimBlock.y));
    // std::cout << "dims " << divup(nx, dimBlock.x) << " " << divup(ny, dimBlock.y) << std::endl;
    mykernel<<<dimGrid, dimBlock>>>(rGPU, dGPU, nx, ny);
    CHECK(hipGetLastError());
    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(result, rGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}